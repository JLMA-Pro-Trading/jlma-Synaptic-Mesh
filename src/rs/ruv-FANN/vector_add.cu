
#include <hip/hip_runtime.h>
// CUDA vector addition kernel
__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

// Matrix multiplication kernel
__global__ void matrixMul(float* a, float* b, float* c, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; k++) {
            sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}